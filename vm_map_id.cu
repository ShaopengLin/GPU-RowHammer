
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

const uint64_t LAYOUT_SIZE = 1048576; // 1073741824; //16106127360;

__global__ void mapping_kernel(uint16_t *target, uint16_t *it_addr, long long *time)
{
  uint16_t buf_t;
  uint16_t buf_it;

  // *it_addr = (uint16_t) it_addr;
  // /* Discard our target's cache from potential previous executions. */
  // asm("discard.global.L2 [%0], 128;" : "+l"(target));

  /* Bring it_addr to its row buffer in advance */
  // asm("ld.u16.global.volatile %0, [%1];" : "=h"(buf_it) : "l"(it_addr));

  // /* If it_addr is in the same bank as target, target will take longer due to conflict*/
  // asm volatile ("ld.u16.global.volatile %0, [%1];" : "=h"(buf_it) : "l"(target));
  asm volatile ("ld.u16.global.cv %0, [%1];" : "=h"(buf_it) : "l"(it_addr));
  asm("discard.global.L2 [%0], 128;" : "+l"(it_addr));
  asm("discard.global.L2 [%0], 128;" : "+l"(target));
  
  asm volatile ("ld.u16.global.volatile %0, [%1];" : "=h"(buf_t) : "l"(target));
  long long start = clock64();
  // asm volatile ("ld.u16.global.volatile %0, [%1];" : "=h"(buf_t) : "l"(target));
  asm volatile ("ld.u16.global.volatile %0, [%1];" : "=h"(buf_it) : "l"(it_addr));
  
  long long end = clock64();

  assert(buf_t == 0);
  assert(buf_it == 0);
  *time = end - start;
}
int main(void)
{
  uint16_t *d_x;
  hipMalloc(&d_x, LAYOUT_SIZE);

  long long *time;
  hipHostAlloc(&time, sizeof(long long), hipHostMallocDefault);
  
  for (int i = 1; i < 20; i++)
  {
    mapping_kernel<<<1, 1>>>(d_x, d_x + i, time);
    hipDeviceSynchronize();
    std::cout << *time << '\n';
  }
  // mapping_kernel<<<1, 1>>>(d_x, d_x, time);
  // cudaDeviceSynchronize();
  // std::cout << *time << '\n';
  size_t f, t;
  hipMemGetInfo(&f, &t);
  std::cout << f << '\n';
  std::cout << t << '\n';
  std::cout << d_x << '\n';
  hipFree(d_x);
}