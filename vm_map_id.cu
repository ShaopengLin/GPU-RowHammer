
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

const uint64_t LAYOUT_SIZE = 16106127360; // 1073741824; //16106127360 // 1048576 // 4294967296;

__global__ void mapping_kernel(uint64_t *target, uint64_t *it_addr, long long *time)
{
  uint64_t buf_t;
  uint64_t buf_it;

  asm volatile ("discard.global.L2 [%0], 128;" : "+l"(target));
  
  long long start = clock64();
  asm volatile ("ld.u64.global.volatile %0, [%1];" : "=l"(buf_t) : "l"(target));
  long long end = clock64();

  *time = end - start;
}
int main(void)
{
  uint64_t *d_x;
  hipMalloc(&d_x, LAYOUT_SIZE);

  long long *time;
  hipHostAlloc(&time, sizeof(long long), hipHostMallocDefault);
  long long max = 0;
  for (int i = 1; i < 200; i++)
  {
      mapping_kernel<<<1, 1>>>(d_x, d_x + i, time);
      hipDeviceSynchronize();
      std::cout << *time << '\n';
  }
  //std::cout << max << '\n';
  // mapping_kernel<<<1, 1>>>(d_x, d_x, time);
  // cudaDeviceSynchronize();
  // std::cout << *time << '\n';
  size_t f, t;
  hipMemGetInfo(&f, &t);
  //std::cout << f << '\n';
  //std::cout << t << '\n';
  //std::cout << d_x << '\n';
  hipFree(d_x);
}